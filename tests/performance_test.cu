#include "hip/hip_runtime.h"
/*
 *  Copyright 2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cuda_common.h"
#include "../int_fastdiv.h"

template<typename divisor_type>
__global__ void throughput_test(
	divisor_type d1,
	divisor_type d2,
	divisor_type d3,
	int dummy,
	int * buf)
{
	int elem_id = blockIdx.x * blockDim.x + threadIdx.x;

	int x1 = elem_id / d1;
	int x2 = elem_id / d2;
	int x3 = elem_id / d3;

	int aggregate = x1 + x2 + x3;	
	if (aggregate & dummy == 1)
		buf[0] = aggregate;
}

template<typename divisor_type>
__global__ void latency_test(
	divisor_type d1,
	divisor_type d2,
	divisor_type d3,
	divisor_type d4,
	divisor_type d5,
	divisor_type d6,
	divisor_type d7,
	divisor_type d8,
	divisor_type d9,
	divisor_type d10,
	int dummy,
	int * buf)
{
	int elem_id = blockIdx.x * blockDim.x + threadIdx.x;

	int x = elem_id;
	x = x / d1;
	x = x / d2;
	x = x / d3;
	x = x / d4;
	x = x / d5;
	x = x / d6;
	x = x / d7;
	x = x / d8;
	x = x / d9;
	x = x / d10;

	if (x & dummy == 1)
		buf[0] = x;
}

int main(int argc, char* argv[])
{
	int grid_size = 32 * 1024;
	int threadblock_size = 256;

	hipEvent_t start, stop;
	float elapsed_time_slow;
	float elapsed_time_fast;

	cuda_safe_call(hipEventCreate(&start));
	cuda_safe_call(hipEventCreate(&stop));

	{
		std::cout << "THROUGHPUT TEST" << std::endl;

		std::cout << "Benchmarking plain division by constant... ";
		cuda_safe_call(hipEventRecord(start, 0));
		throughput_test<int><<<grid_size, threadblock_size>>>(3, 5, 7, 0, 0);
		cuda_safe_call(hipEventRecord(stop, 0));
		cuda_safe_call(hipEventSynchronize(stop));
		cuda_safe_call(hipEventElapsedTime(&elapsed_time_slow, start, stop));
		std::cout << elapsed_time_slow << " milliseconds" << std::endl;

		std::cout << "Benchmarking fast division by constant... ";
		cuda_safe_call(hipEventRecord(start, 0));
		throughput_test<int_fastdiv><<<grid_size, threadblock_size>>>(3, 5, 7, 0, 0);
		cuda_safe_call(hipEventRecord(stop, 0));
		cuda_safe_call(hipEventSynchronize(stop));
		cuda_safe_call(hipEventElapsedTime(&elapsed_time_fast, start, stop));
		std::cout << elapsed_time_fast << " milliseconds" << std::endl;

		std::cout << "Speedup = " << elapsed_time_slow / elapsed_time_fast << std::endl;
	}

	{
		std::cout << "LATENCY TEST" << std::endl;

		std::cout << "Benchmarking plain division by constant... ";
		cuda_safe_call(hipEventRecord(start, 0));
		latency_test<int><<<grid_size, threadblock_size>>>(1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 0, 0);
		cuda_safe_call(hipEventRecord(stop, 0));
		cuda_safe_call(hipEventSynchronize(stop));
		cuda_safe_call(hipEventElapsedTime(&elapsed_time_slow, start, stop));
		std::cout << elapsed_time_slow << " milliseconds" << std::endl;

		std::cout << "Benchmarking fast division by constant... ";
		cuda_safe_call(hipEventRecord(start, 0));
		latency_test<int_fastdiv><<<grid_size, threadblock_size>>>(1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 0, 0);
		cuda_safe_call(hipEventRecord(stop, 0));
		cuda_safe_call(hipEventSynchronize(stop));
		cuda_safe_call(hipEventElapsedTime(&elapsed_time_fast, start, stop));
		std::cout << elapsed_time_fast << " milliseconds" << std::endl;

		std::cout << "Speedup = " << elapsed_time_slow / elapsed_time_fast << std::endl;
	}

	cuda_safe_call(hipEventDestroy(start));
	cuda_safe_call(hipEventDestroy(stop));

	return 0;
}
